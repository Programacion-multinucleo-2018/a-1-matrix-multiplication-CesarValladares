#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cstdio>
#include <chrono>
#include "common.h"

using namespace std;

#define SIZEM 1000;

void fillMatrices(float * ip, const int size){

    int i; 

    for (i = 0; i < size; i++){
        ip[i] = i;
    }    
}

__global__ void multMatrixOnGPU2D(float *MatA, float *MatB, float *MatC, int nx,
    int ny)
{   
    unsigned int ix = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int iy = threadIdx.y + blockIdx.y * blockDim.y;

    unsigned int idx = ix * nx + iy;

    float auxiliar = 0.0;

    if (ix < nx && iy < ny){
        for(int i = 0; i < ny ; i++){
            auxiliar += MatA[ix * nx + i] * MatB[i * ny + iy];
        }
    }

    MatC[idx] = auxiliar;
}

int main (int argc, char ** argv){

    // Set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    SAFE_CALL(hipGetDeviceProperties(&deviceProp, dev), "Error device prop");
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    SAFE_CALL(hipSetDevice(dev), "Error setting device");

    // Tamaño de la matriz
    int nx = SIZEM;
    int ny = SIZEM;

    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);
    printf("Matrix size: nx %d ny %d\n", nx, ny);

    // Apartar memoria 
    float *h_A, *h_B, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // Inicializar matrices
    fillMatrices(h_A, nxy);
    fillMatrices(h_B, nxy);

    memset(gpuRef, 0, nBytes);

    // Apartar memoria en la GPU
    float *d_MatA, *d_MatB, *d_MatC;
    SAFE_CALL(hipMalloc((void **)&d_MatA, nBytes), "Error allocating d_MatA");
    SAFE_CALL(hipMalloc((void **)&d_MatB, nBytes), "Error allocating d_MatB");
    SAFE_CALL(hipMalloc((void **)&d_MatC, nBytes), "Error allocating d_MatC");

    // Transferir informacion a la GPU
    SAFE_CALL(hipMemcpy(d_MatA, h_A, nBytes, hipMemcpyHostToDevice), "Error copying d_MatA");
    SAFE_CALL(hipMemcpy(d_MatB, h_B, nBytes, hipMemcpyHostToDevice), "Error copying d_MatB");

    // Invocar al kernel del lado del host
    int dimx = 64;
    int dimy = 16;
    dim3 block(dimx, dimy);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    int repetitions = 100;
    auto average = 0;

    for (int i = 0; i < repetitions; i++){
        auto start_cpu =  chrono::high_resolution_clock::now();
        multMatrixOnGPU2D<<<grid, block>>>(d_MatA, d_MatB, d_MatC, nx, ny);
        SAFE_CALL(hipDeviceSynchronize(), "Error executing kernel");
        auto end_cpu =  chrono::high_resolution_clock::now();
        
        chrono::duration<float, std::milli> duration_ms = end_cpu - start_cpu;
        average += duration_ms.count();
    }

    average /= repetitions;

    printf("multMatrixOnGPU1D <<<(%d,%d), (%d,%d)>>> elapsed %d ms in %d repetitions\n", grid.x,
           grid.y,
           block.x, block.y, average, repetitions);

    // SAFE_CALL kernel error
    SAFE_CALL(hipGetLastError(), "Error with last error");

    // copy kernel result back to host side
    SAFE_CALL(hipMemcpy(gpuRef, d_MatC, nBytes, hipMemcpyDeviceToHost), "Error copying d_MatC");

    // free device global memory
    SAFE_CALL(hipFree(d_MatA), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatB), "Error freeing memory");
    SAFE_CALL(hipFree(d_MatC), "Error freeing memory");

    // free host memory
    free(h_A);
    free(h_B);
    free(gpuRef);

    // reset device
    SAFE_CALL(hipDeviceReset(), "Error reseting");

    return 0;
}